#include <iostream>
#include <argparse/argparse.hpp>

#include "graph/graph.h"
#include "tc/tc.cuh"

#include <loguru.hpp>
//#define LOGURU_WITH_STREAMS 1


int main(int argc, char* argv[]) {
    loguru::init(argc, argv);
    LOG_F(INFO, "Hello from main.cpp!");
    LOG_F(INFO, "main function about to end!");

    argparse::ArgumentParser parser("triangle counting");

    parser.add_argument("--gpu")
            .help("GPU Device ID (must be a positive integer)")
            .default_value(0)
            .action([](const std::string &value) { return std::stoi(value); });

    parser.add_argument("--graph")
            .help("Graph file path")
            .default_value("/")
            .action([](const std::string &value) { return value; });

    try {
        parser.parse_args(argc, argv);
    } catch (const std::exception& err) {
        std::cout << parser << std::endl;
        exit(EXIT_FAILURE);
    }

    auto device_count = 0;
    auto device_id = 0;

    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        std::cerr << "error: no gpu device found" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (parser.is_used("--gpu")) {
        device_id = parser.get<int>("--gpu");
        if (device_id >= device_count) {
            std::cerr << "error: invalid gpu device id" << std::endl;
            exit(EXIT_FAILURE);
        }
        hipSetDevice(device_id);
    }

    if (parser.is_used("--graph")) {
        auto dataset = parser.get<std::string>("--graph");
        auto g = Graph(dataset);

        // then aglorithm
        tc(&g);
    }

}
